#include "hip/hip_runtime.h"
// Original author: Adam Polak
// 
// edited by: Younes Ouazref
// 

#include "gpu.h"

#include "gpu-thrust.h"
#include "timer.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

template<bool ZIPPED>
__global__ void CalculateNodePointers(int n, int m, int* edges, int* nodes) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i <= m; i += step) {
    int prev = i > 0 ? edges[ZIPPED ? (2 * (i - 1) + 1) : (m + i - 1)] : -1;
    int next = i < m ? edges[ZIPPED ? (2 * i + 1) : (m + i)] : n;
    for (int j = prev + 1; j <= next; ++j)
      nodes[j] = i;
  }
}

__global__ void CalculateFlags(int m, int* edges, int* nodes, bool* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = nodes[a + 1] - nodes[a];
    int deg_b = nodes[b + 1] - nodes[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}

__global__ void CalculateTriangles(
    int m, const int* __restrict__ edges, const int* __restrict__ nodes,
    uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
  int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  uint64_t count = 0;

  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = nodes[u], u_end = nodes[u + 1];
    int v_it = nodes[v], v_end = nodes[v + 1];

    int a = edges[u_it], b = edges[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges[++u_it];
      if (d >= 0)
        b = edges[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}

Edges RemoveBackwardEdgesCPU(const Edges& unordered_edges) {
  int n = NumVertices(unordered_edges);
  int m = unordered_edges.size();

  vector<int> deg(n);
  for (int i = 0; i < m; ++i)
    ++deg[unordered_edges[i].first];

  vector< pair<int, int> > edges;
  edges.reserve(m / 2);
  for (int i = 0; i < m; ++i) {
    int s = unordered_edges[i].first, t = unordered_edges[i].second;
    if (deg[s] > deg[t] || (deg[s] == deg[t] && s > t))
      edges.push_back(make_pair(s, t));
  }

  return edges;
}

uint64_t MultiGPUCalculateTriangles(
    int n, int m, int* dev_edges, int* dev_nodes, int device_count) {
  vector<int*> multi_dev_edges(device_count);
  vector<int*> multi_dev_nodes(device_count);

  multi_dev_edges[0] = dev_edges;
  multi_dev_nodes[0] = dev_nodes;

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipMalloc(&multi_dev_edges[i], m * 2 * sizeof(int)));
    CUCHECK(hipMalloc(&multi_dev_nodes[i], (n + 1) * sizeof(int)));
    int dst = i, src = (i + 1) >> 2;
    CUCHECK(hipMemcpyPeer(
          multi_dev_edges[dst], dst, multi_dev_edges[src], src,
          m * 2 * sizeof(int)));
    CUCHECK(hipMemcpyPeer(
          multi_dev_nodes[dst], dst, multi_dev_nodes[src], src,
          (n + 1) * sizeof(int)));
  }

  vector<int> NUM_BLOCKS(device_count);
  vector<uint64_t*> multi_dev_results(device_count);

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    NUM_BLOCKS[i] = NUM_BLOCKS_PER_MP * NumberOfMPs();
    CUCHECK(hipMalloc(
          &multi_dev_results[i],
          NUM_BLOCKS[i] * NUM_THREADS * sizeof(uint64_t)));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1));
    CalculateTriangles<<<NUM_BLOCKS[i], NUM_THREADS>>>(
        m, multi_dev_edges[i], multi_dev_nodes[i], multi_dev_results[i],
        device_count, i);
  }

  uint64_t result = 0;

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipDeviceSynchronize());
    result += SumResults(NUM_BLOCKS[i] * NUM_THREADS, multi_dev_results[i]);
  }

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_edges[i]));
    CUCHECK(hipFree(multi_dev_nodes[i]));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_results[i]));
  }

  hipSetDevice(0);
  return result;
}

uint64_t GpuForward(const Edges& edges) {
  return MultiGpuForward(edges, 1);
}

uint64_t MultiGpuForward(const Edges& edges, int device_count) {
  Timer* timer = Timer::NewTimer();

  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  int m = edges.size(), n;

  int* dev_edges;
  int* dev_nodes;

  // Creation of the cuda events which will count the GPU execution time.
  // -----------------------------
  hipEvent_t Malloc1Start, Malloc1Stop;
  hipEventCreate(&Malloc1Start);
  hipEventCreate(&Malloc1Stop);
  // -----------------------------
  // ------------------------------
  hipEvent_t Memcpy1Start, Memcpy1Stop;
  hipEventCreate(&Memcpy1Start);
  hipEventCreate(&Memcpy1Stop);
  // -------------------------------
  // -----------------------------------------
  hipEvent_t GPUthrustVerticesStart, GPUthrustVerticesStop;
  hipEventCreate(&GPUthrustVerticesStart);
  hipEventCreate(&GPUthrustVerticesStop);
  // ------------------------------------------
  // --------------------------------------
  hipEvent_t GPUthrustSortStart, GPUthrustSortStop;
  hipEventCreate(&GPUthrustSortStart);
  hipEventCreate(&GPUthrustSortStop);
  // --------------------------------------
  // ------------------------------
  hipEvent_t Malloc2Start, Malloc2Stop;
  hipEventCreate(&Malloc2Start);
  hipEventCreate(&Malloc2Stop);
  // -------------------------------
  // --------------------------------
  hipEvent_t CalcNodeP_TStart, CalcNodeP_TStop;
  hipEventCreate(&CalcNodeP_TStart);
  hipEventCreate(&CalcNodeP_TStop);
  // ---------------------------------
  // -----------------------------
  hipEvent_t Malloc3Start, Malloc3Stop;
  hipEventCreate(&Malloc3Start);
  hipEventCreate(&Malloc3Stop);
  // -----------------------------
  // ---------------------------------
  hipEvent_t CalcFlagsStart, CalcFlagsStop;
  hipEventCreate(&CalcFlagsStart);
  hipEventCreate(&CalcFlagsStop);
  // ----------------------------------
  // ---------------------------------------
  hipEvent_t GPUthrustRemoveStart, GPUthrustRemoveStop;
  hipEventCreate(&GPUthrustRemoveStart);
  hipEventCreate(&GPUthrustRemoveStop);
  // ----------------------------------------
  // --------------------------
  hipEvent_t UnzipStart, UnzipStop;
  hipEventCreate(&UnzipStart);
  hipEventCreate(&UnzipStop);
  // --------------------------
  // -------------------------------
  hipEvent_t Memcpy2Start, Memcpy2Stop;
  hipEventCreate(&Memcpy2Start);
  hipEventCreate(&Memcpy2Stop);
  // --------------------------------
  // ------------------------------
  hipEvent_t CalcNodeP_FStart, CalcNodeP_FStop;
  hipEventCreate(&CalcNodeP_FStart);
  hipEventCreate(&CalcNodeP_FStop);
  // -------------------------------
  // ----------------------------
  hipEvent_t Malloc4Start, Malloc4Stop;
  hipEventCreate(&Malloc4Start);
  hipEventCreate(&Malloc4Stop);
  // ---------------------------
  // ------------------------
  hipEvent_t TCountStart, TCountStop;
  hipEventCreate(&TCountStart);
  hipEventCreate(&TCountStop);
  // -------------------------
  // -------------------------------
  hipEvent_t GPUthrustSumStart, GPUthrustSumStop;
  hipEventCreate(&GPUthrustSumStart);
  hipEventCreate(&GPUthrustSumStop);
  // -------------------------------


  if ((uint64_t)m * 4 * sizeof(int) < GlobalMemory()) {  // just approximation


    hipEventRecord(Malloc1Start, 0);
    CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
    hipEventRecord(Malloc1Stop, 0);

    

    hipEventRecord(Memcpy1Start, 0);
    CUCHECK(hipMemcpyAsync(
          dev_edges, edges.data(), m * 2 * sizeof(int),
          hipMemcpyHostToDevice));
    hipEventRecord(Memcpy1Stop, 0);

    
    CUCHECK(hipDeviceSynchronize());
    // timer->Done("Memcpy edges from host do device");



    hipEventRecord(GPUthrustVerticesStart, 0);
    n = NumVerticesGPU(m, dev_edges);
    hipEventRecord(GPUthrustVerticesStop, 0);

    cout << "Num vertices: " << n << endl;
    // Undirected graph.
    cout << "Num edges: " << (m/2) << endl;


    // timer->Done("Calculate number of vertices");
    hipEventRecord(GPUthrustSortStart, 0);
    SortEdges(m, dev_edges);
    hipEventRecord(GPUthrustSortStop, 0);

    CUCHECK(hipDeviceSynchronize());
    // timer->Done("Sort edges");



    hipEventRecord(Malloc2Start, 0);
    CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
    hipEventRecord(Malloc2Start, 0);



    hipEventRecord(CalcNodeP_TStart, 0);
    CalculateNodePointers<true><<<NUM_BLOCKS, NUM_THREADS>>>(
        n, m, dev_edges, dev_nodes);
    hipEventRecord(CalcNodeP_TStop, 0);


    CUCHECK(hipDeviceSynchronize());
    // timer->Done("Calculate nodes array for two-way zipped edges");

    bool* dev_flags;



    hipEventRecord(Malloc3Start, 0);
    CUCHECK(hipMalloc(&dev_flags, m * sizeof(bool)));
    hipEventRecord(Malloc3Stop, 0);



    hipEventRecord(CalcFlagsStart, 0);
    CalculateFlags<<<NUM_BLOCKS, NUM_THREADS>>>(
        m, dev_edges, dev_nodes, dev_flags);
    hipEventRecord(CalcFlagsStop, 0);



    hipEventRecord(GPUthrustRemoveStart, 0);
    RemoveMarkedEdges(m, dev_edges, dev_flags);
    hipEventRecord(GPUthrustRemoveStop, 0);


    CUCHECK(hipFree(dev_flags));
    CUCHECK(hipDeviceSynchronize());
    m /= 2;
    // timer->Done("Remove backward edges");


    hipEventRecord(UnzipStart, 0);
    UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_edges, dev_edges + 2 * m);
    hipEventRecord(UnzipStop, 0);


    hipEventRecord(Memcpy2Start, 0);
    CUCHECK(hipMemcpyAsync(
          dev_edges, dev_edges + 2 * m, 2 * m * sizeof(int),
          hipMemcpyDeviceToDevice));
    hipEventRecord(Memcpy2Stop, 0);


    CUCHECK(hipDeviceSynchronize());
    // timer->Done("Unzip edges");


  } else {
    Edges fwd_edges = RemoveBackwardEdgesCPU(edges);
    m /= 2;
    timer->Done("Remove backward edges on CPU");

    int* dev_temp;
    CUCHECK(hipMalloc(&dev_temp, m * 2 * sizeof(int)));
    CUCHECK(hipMemcpyAsync(
          dev_temp, fwd_edges.data(), m * 2 * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
    timer->Done("Memcpy edges from host do device");

    SortEdges(m, dev_temp);
    CUCHECK(hipDeviceSynchronize());
    timer->Done("Sort edges");

    CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
    UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_temp, dev_edges);
    CUCHECK(hipFree(dev_temp));
    CUCHECK(hipDeviceSynchronize());
    timer->Done("Unzip edges");

    n = NumVerticesGPU(m, dev_edges);
    CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
    timer->Done("Calculate number of vertices");
  }


  hipEventRecord(CalcNodeP_FStart, 0);
  CalculateNodePointers<false><<<NUM_BLOCKS, NUM_THREADS>>>(
      n, m, dev_edges, dev_nodes);
  hipEventRecord(CalcNodeP_FStop, 0);



  CUCHECK(hipDeviceSynchronize());
  // timer->Done("Calculate nodes array for one-way unzipped edges");

  uint64_t result = 0;




  if (device_count == 1) {
    uint64_t* dev_results;


    hipEventRecord(Malloc4Start, 0);
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    hipEventRecord(Malloc4Stop, 0);


    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1);


    hipProfilerStart();

    hipEventRecord(TCountStart, 0);
    CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
        m, dev_edges, dev_nodes, dev_results);
    hipEventRecord(TCountStop, 0);

    CUCHECK(hipDeviceSynchronize());
    
    hipProfilerStop();
    // timer->Done("Calculate triangles");


    hipEventRecord(GPUthrustSumStart, 0);
    result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    hipEventRecord(GPUthrustSumStop, 0);


    CUCHECK(hipFree(dev_results));



    // timer->Done("Reduce");
  } else {
    result = MultiGPUCalculateTriangles(
        n, m, dev_edges, dev_nodes, device_count);
    timer->Done("Calculate triangles on multi GPU");
  }

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_nodes));

  delete timer;






  CUCHECK(hipEventSynchronize(Malloc1Stop));
  // print the time the kernel invocation took, without the copies!
  float Malloc1ElapsedTime;
  hipEventElapsedTime(&Malloc1ElapsedTime, Malloc1Start, Malloc1Stop);
  cout << "Malloc1 copying took: " << Malloc1ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Malloc1Start);
  hipEventDestroy(Malloc1Stop);


  CUCHECK(hipEventSynchronize(Memcpy1Stop));
  // print the time the kernel invocation took, without the copies!
  float Memcpy1ElapsedTime;
  hipEventElapsedTime(&Memcpy1ElapsedTime, Memcpy1Start, Memcpy1Stop);
  cout << "Memcpy1 took: " << Memcpy1ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Memcpy1Start);
  hipEventDestroy(Memcpy1Stop);

  
  CUCHECK(hipEventSynchronize(GPUthrustVerticesStop));
  // print the time the kernel invocation took, without the copies!
  float GPUthrustVerticesElapsedTime;
  hipEventElapsedTime(&GPUthrustVerticesElapsedTime, GPUthrustVerticesStart, GPUthrustVerticesStop);
  cout << "GPU thrust num of vertices took: " << GPUthrustVerticesElapsedTime << " milliseconds" << endl;
  hipEventDestroy(GPUthrustVerticesStart);
  hipEventDestroy(GPUthrustVerticesStop);


  CUCHECK(hipEventSynchronize(GPUthrustSortStop));
  // print the time the kernel invocation took, without the copies!
  float GPUthrustSortElapsedTime;
  hipEventElapsedTime(&GPUthrustSortElapsedTime, GPUthrustSortStart, GPUthrustSortStop);
  cout << "GPU thrust sort took: " << GPUthrustSortElapsedTime << " milliseconds" << endl;
  hipEventDestroy(GPUthrustSortStart);
  hipEventDestroy(GPUthrustSortStop);


  CUCHECK(hipEventSynchronize(Malloc2Stop));
  // print the time the kernel invocation took, without the copies!
  float Malloc2ElapsedTime;
  hipEventElapsedTime(&Malloc2ElapsedTime, Malloc2Start, Malloc2Stop);
  cout << "Malloc2 copying took: " << Malloc2ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Malloc2Start);
  hipEventDestroy(Malloc2Stop);


  CUCHECK(hipEventSynchronize(CalcNodeP_TStop));
  // print the time the kernel invocation took, without the copies!
  float CalcNodeP_TElapsedTime;
  hipEventElapsedTime(&CalcNodeP_TElapsedTime, CalcNodeP_TStart, CalcNodeP_TStop);
  cout << "Calc node pointers T took: " << CalcNodeP_TElapsedTime << " milliseconds" << endl;
  hipEventDestroy(CalcNodeP_TStart);
  hipEventDestroy(CalcNodeP_TStop);


  CUCHECK(hipEventSynchronize(Malloc3Stop));
  // print the time the kernel invocation took, without the copies!
  float Malloc3ElapsedTime;
  hipEventElapsedTime(&Malloc3ElapsedTime, Malloc3Start, Malloc3Stop);
  cout << "Malloc3 took: " << Malloc3ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Malloc3Start);
  hipEventDestroy(Malloc3Stop);


  CUCHECK(hipEventSynchronize(CalcFlagsStop));
  // print the time the kernel invocation took, without the copies!
  float CalcFlagsElapsedTime;
  hipEventElapsedTime(&CalcFlagsElapsedTime, CalcFlagsStart, CalcFlagsStop);
  cout << "Calc flags took: " << CalcFlagsElapsedTime << " milliseconds" << endl;
  hipEventDestroy(CalcFlagsStart);
  hipEventDestroy(CalcFlagsStop);

  CUCHECK(hipEventSynchronize(GPUthrustRemoveStop));
  // print the time the kernel invocation took, without the copies!
  float GPUthrustRemoveElapsedTime;
  hipEventElapsedTime(&GPUthrustRemoveElapsedTime, GPUthrustRemoveStart, GPUthrustRemoveStop);
  cout << "GPU thrust remove took: " << GPUthrustRemoveElapsedTime << " milliseconds" << endl;
  hipEventDestroy(GPUthrustRemoveStart);
  hipEventDestroy(GPUthrustRemoveStop);


  CUCHECK(hipEventSynchronize(UnzipStop));
  // print the time the kernel invocation took, without the copies!
  float UnzipElapsedTime;
  hipEventElapsedTime(&UnzipElapsedTime, UnzipStart, UnzipStop);
  cout << "Unzipping took: " << UnzipElapsedTime << " milliseconds" << endl;
  hipEventDestroy(UnzipStart);
  hipEventDestroy(UnzipStop);

  CUCHECK(hipEventSynchronize(Memcpy2Stop));
  // print the time the kernel invocation took, without the copies!
  float Memcpy2ElapsedTime;
  hipEventElapsedTime(&Memcpy2ElapsedTime, Memcpy2Start, Memcpy2Stop);
  cout << "Memcpy2 took: " << Memcpy2ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Memcpy2Start);
  hipEventDestroy(Memcpy2Stop);


  CUCHECK(hipEventSynchronize(CalcNodeP_FStop));
  // print the time the kernel invocation took, without the copies!
  float CalcNodeP_FElapsedTime;
  hipEventElapsedTime(&CalcNodeP_FElapsedTime, CalcNodeP_FStart, CalcNodeP_FStop);
  cout << "Calc node pointers F took: " << CalcNodeP_TElapsedTime << " milliseconds" << endl;
  hipEventDestroy(CalcNodeP_FStart);
  hipEventDestroy(CalcNodeP_FStop);


  CUCHECK(hipEventSynchronize(Malloc4Stop));
  // print the time the kernel invocation took, without the copies!
  float Malloc4ElapsedTime;
  hipEventElapsedTime(&Malloc4ElapsedTime, Malloc4Start, Malloc4Stop);   
  cout << "Malloc4 took: " << Malloc4ElapsedTime << " milliseconds" << endl;
  hipEventDestroy(Malloc4Start);
  hipEventDestroy(Malloc4Stop);


  CUCHECK(hipEventSynchronize(TCountStop));
  // print the time the kernel invocation took, without the copies!
  float TCountElapsedTime;
  hipEventElapsedTime(&TCountElapsedTime, TCountStart, TCountStop);   
  cout << "Triangle count took: " << TCountElapsedTime << " milliseconds" << endl;
  hipEventDestroy(TCountStart);
  hipEventDestroy(TCountStop);


  CUCHECK(hipEventSynchronize(GPUthrustSumStop));
  // print the time the kernel invocation took, without the copies!
  float GPUthrustSumElapsedTime;
  hipEventElapsedTime(&GPUthrustSumElapsedTime, GPUthrustSumStart, GPUthrustSumStop);   
  cout << "Sum of triangles (of threads) took: " << GPUthrustSumElapsedTime << " milliseconds" << endl;
  hipEventDestroy(GPUthrustSumStart);
  hipEventDestroy(GPUthrustSumStop);

  // The output of the cuda events are being saved by the script that runs them.
  // the ':' char is used to recognize the data.
  cout << "Total GPU time: " << GPUthrustSumElapsedTime + TCountElapsedTime + Malloc4ElapsedTime + 
      CalcNodeP_TElapsedTime + Memcpy2ElapsedTime + UnzipElapsedTime + GPUthrustRemoveElapsedTime + CalcFlagsElapsedTime
      + Malloc3ElapsedTime + CalcNodeP_TElapsedTime + Malloc2ElapsedTime + GPUthrustSortElapsedTime
      + GPUthrustVerticesElapsedTime + Memcpy1ElapsedTime + Malloc1ElapsedTime << " milliseconds" << endl;


  return result;
}

void PreInitGpuContext(int device) {
  CUCHECK(hipSetDevice(device));
  CUCHECK(hipFree(NULL));
}
