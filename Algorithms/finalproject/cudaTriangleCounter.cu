#include "hip/hip_runtime.h"
/*
 * Triangle counter without workload balancing
 *
 * @author: Manish Jain
 * @author: Vashishtha Adtani
 *
 * edited by: Younes Ouazref 
 *
 */

#include <iostream>
#include <string>
#include <sstream>
#include <algorithm>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>                                                        
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cudaTriangleCounter.h"

#define BLOCK_SIZE 112

using namespace std;


float firstAllocation = 0;


struct GlobalConstants {

    int *NodeList;
    int *ListLen;
    int numNodes;
    int numEdges;
};

__constant__ GlobalConstants cuConstCounterParams;

void
CudaTriangleCounter::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    // printf("---------------------------------------------------------\n");
    // printf("Initializing CUDA for CountingTriangles\n");
    // printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        // printf("Device %d: %s\n", i, deviceProps.name);
        // printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        // printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        // printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);

    }
    // printf("---------------------------------------------------------\n");

    // By this time the graph should be loaded.  Copying graph to 
    // data structures into device memory so that it is accessible to
    // CUDA kernels
    //


    // Create cuda events to keep track of the GPU time
    hipEvent_t Malloc1Start, Malloc1Stop;
    hipEventCreate(&Malloc1Start);
    hipEventCreate(&Malloc1Stop);
    // ----------------------------
    hipEvent_t Memcpy1Start, Memcpy1Stop;
    hipEventCreate(&Memcpy1Start);
    hipEventCreate(&Memcpy1Stop);
    // ----------------------------
    hipEvent_t Malloc2Start, Malloc2Stop;
    hipEventCreate(&Malloc2Start);
    hipEventCreate(&Malloc2Stop);
    // ----------------------------
    hipEvent_t Memcpy2Start, Memcpy2Stop;
    hipEventCreate(&Memcpy2Start);
    hipEventCreate(&Memcpy2Stop);
    // ----------------------------
    hipEvent_t MemcpyToS1Start, MemcpyToS1Stop;
    hipEventCreate(&MemcpyToS1Start);
    hipEventCreate(&MemcpyToS1Stop);
    // ----------------------------

    hipEventRecord(Malloc1Start, 0);
    hipMalloc(&cudaDeviceListLen, sizeof(int ) * numNodes);
    hipEventRecord(Malloc1Stop, 0);

    hipEventRecord(Memcpy1Start, 0);
    hipMemcpy(cudaDeviceListLen, list_len, sizeof(int) * numNodes, hipMemcpyHostToDevice);
    hipEventRecord(Memcpy1Stop, 0);

    hipEventRecord(Malloc2Start, 0);
    hipMalloc((void **)&cudaDeviceNodeList, node_list_size * sizeof(int));
    hipEventRecord(Malloc2Stop, 0);

    hipEventRecord(Memcpy2Start, 0);
    hipMemcpy(cudaDeviceNodeList, node_list, sizeof(int) * node_list_size, hipMemcpyHostToDevice);
    hipEventRecord(Memcpy2Stop, 0);


    GlobalConstants params;
    params.ListLen = cudaDeviceListLen;
    params.NodeList = cudaDeviceNodeList;
    params.numNodes = numNodes;
    params.numEdges = numEdges;


    cout << "Num vertices: " << numNodes << endl;
    cout << "Num edges: " << numEdges << endl;


    hipEventRecord(MemcpyToS1Start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstCounterParams), &params, sizeof(GlobalConstants));
    hipEventRecord(MemcpyToS1Stop, 0);


    hipEventSynchronize(Malloc1Stop);
    float Malloc1ElapsedTime;
    hipEventElapsedTime(&Malloc1ElapsedTime, Malloc1Start, Malloc1Stop);
    cout << "Malloc1 copying took: " << Malloc1ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Malloc1Start);
    hipEventDestroy(Malloc1Stop);

    hipEventSynchronize(Memcpy1Stop);
    float Memcpy1ElapsedTime;
    hipEventElapsedTime(&Memcpy1ElapsedTime, Memcpy1Start, Memcpy1Stop);
    cout << "Memcpy1 copying took: " << Memcpy1ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Memcpy1Start);
    hipEventDestroy(Memcpy1Stop);


    hipEventSynchronize(Malloc2Stop);
    float Malloc2ElapsedTime;
    hipEventElapsedTime(&Malloc2ElapsedTime, Malloc2Start, Malloc2Stop);
    cout << "Malloc2 copying took: " << Malloc2ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Malloc2Start);
    hipEventDestroy(Malloc2Stop);


    hipEventSynchronize(Memcpy2Stop);
    float Memcpy2ElapsedTime;
    hipEventElapsedTime(&Memcpy2ElapsedTime, Memcpy2Start, Memcpy2Stop);
    cout << "Memcpy2 copying took: " << Memcpy2ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Memcpy2Start);
    hipEventDestroy(Memcpy2Stop);


    hipEventSynchronize(MemcpyToS1Stop);
    float MemcpyToS1ElapsedTime;
    hipEventElapsedTime(&MemcpyToS1ElapsedTime, MemcpyToS1Start, MemcpyToS1Stop);
    cout << "MemcpyToS1 copying took: " << MemcpyToS1ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(MemcpyToS1Start);
    hipEventDestroy(MemcpyToS1Stop);


    // Save the time to use at the bottom of the file
    firstAllocation = Malloc1ElapsedTime + Memcpy1ElapsedTime + Malloc2ElapsedTime + Memcpy2ElapsedTime + MemcpyToS1ElapsedTime;



}

CudaTriangleCounter::CudaTriangleCounter(char *fileName) {
    clock_t start, diff, malloc_diff;
    int node, edge_id, temp = 0;
    int total_nodes = 0;
    int total_edges = 0;
    int msec;

    std::string line;
    std::ifstream myfile;
    myfile.open(fileName);

    std::string token;                                                             
    if (strstr(fileName,"new_orkut") != NULL) {                                    
        printf("This is the NEW_ORKUT FILE **\n");                             
        total_nodes = 3072600;                                                     
        total_edges = 117185083 + 1;                                               
    } else {                                                                       
        std::getline(myfile,line);                                                 
        std::stringstream lineStream(line);                                        
        while (lineStream >> token) {                                              
            if (temp == 0) {                                                       
                total_nodes = std::stoi(token, NULL, 10) + 1;                      
            } else if (temp == 1) {                                                
                total_edges = std::stoi(token, NULL, 10) + 1;                      
            } else {                                                               
                printf("!!!!!!!!!!!! TEMP IS %d\n ", temp);                        
                break;                                                             
            }                                                                      
            temp++;                                                                
        }                                                                          
    }

    start = clock();

    numNodes = total_nodes;
    node_list_size = total_edges * 2;
    numEdges = total_edges;

    // printf("total_nodes %d\n", total_nodes);
    // printf("node_list_size %d\n", node_list_size);
    // printf("numEdges %d\n", numEdges);




    list_len = (int *)calloc(total_nodes, sizeof(int));
    start_addr = (int *)calloc(total_nodes, sizeof(int));
    node_list = (int *)calloc(node_list_size, sizeof(int));

    malloc_diff = clock() - start;
    msec = malloc_diff * 1000 / CLOCKS_PER_SEC;

    // printf("memory allocated ......\n");
    node = 1;
    temp = 1;
    int neighbors;
    while(std::getline(myfile, line)) {
        neighbors = 0;
        std::stringstream lineStream(line);
        std::string token;
        while(lineStream >> token)
        {
            edge_id = std::stoi(token, NULL, 10);
            if (edge_id > node) {
                node_list[temp++] = edge_id;
                neighbors++;
            }
        }

        list_len[node] = neighbors;
        node++;
    }

    // printf("graph created......\n");
    diff = clock() - start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    // printf("time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

    myfile.close();
}

CudaTriangleCounter::~CudaTriangleCounter() {

    free(node_list);
    free(list_len);
}

/************************* Sequential implementation *************************/

#if 0
void CudaTriangleCounter::countTriangles() {
    int i, j, k, m, count=0;

    for (i=1; i<numNodes; i++) {

        int *list = node_list + start_addr[i-1] + 1;

        int len = list_len[i];

        if (len < 2) {
            continue;
        }

        for (j=0; j<len-1; j++) {
            for (k=j+1; k<len; k++) {

                int idx1;
                int idx2;
                idx1 = list[j];
                idx2 = list[k];
                int *list1 = node_list + start_addr[idx1-1] + 1;
                int len1 = list_len[idx1];

                for (m=0; m<len1; m++) {

                    if (list1[m] == idx2) {
                        count++;
                    }
                }
            }

        }

    }
        printf("count for %d -> %d\n", i, count);

}
#endif

/***************** First implementation using vertices to count triangles **********************/
//Performs poorly that's why why we shifted to our final approach mentioned below
#if 0
__global__ void countTriangleKernel(int *countArray) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cuConstCounterParams.numNodes) {
        return;
    }

    int j, k, m, count=0;
    int *node_list = cuConstCounterParams.NodeList;
    int *list_len = cuConstCounterParams.ListLen;
    int *start_addr = cuConstCounterParams.StartAddr;

    int *list = node_list + start_addr[i-1] + 1;
    int len = list_len[i];

    if (len < 2) {
        countArray[i] = 0;
        return;
    }

    for (j=0; j<len-1; j++) {
        for (k=j+1; k<len; k++) {

            int idx1;
            int idx2;
            idx1 = list[j];
            idx2 = list[k];
            int *list1 = node_list + start_addr[idx1-1] + 1;
            int len1 = list_len[idx1];

            for (m=0; m<len1; m++) {
                if (list1[m] == idx2) {
                    count++;
                }
            }
        }
    }

    countArray[i] = count;

   //printf("%d count %d\n", i, count);
}

void
CudaTriangleCounter::countTriangles() {

    dim3 blockdim  = 1024;
    dim3 griddim = (numNodes + 1024)/1024;
    int *countArray;
    int count;

    hipMalloc((void **)&countArray, numNodes * sizeof(int));

    printf("countTriangleKernel\n");
    countTriangleKernel<<<griddim, blockdim>>>(countArray);
    hipDeviceSynchronize();
    printf("countTriangleKernel done\n");

    thrust::device_ptr<int> dev_ptr(countArray);
    thrust::inclusive_scan(dev_ptr, dev_ptr + numNodes, dev_ptr);

    hipMemcpy(&count, &countArray[numNodes-1], sizeof(int), hipMemcpyDeviceToHost);

    printf("count %d\n", count);
}

#endif

/************** Final approach but without work load balancing *************/

/*
 * Kernel to count number of triangles formed by a single edge. And store the count
 * in an array on which we will run reduction later to find total number of triangles
 * in the given graph.
 */
__global__ void countTriangleKernel(int *countArray, edge_tuple_t *compressed_list, int *start_addr) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cuConstCounterParams.numEdges) {
        return;
    }

    if (i == 0) {
        countArray[i] = 0;
        return;
    }

    int j = 0, k = 0;
    uint64_t count=0;
    int *node_list = cuConstCounterParams.NodeList;
    int *list_len = cuConstCounterParams.ListLen;
    edge_tuple_t *edgeList = compressed_list;

    int u = edgeList[i].u;
    int v = edgeList[i].v;

    /* Fetching neigbour vertices from the node list */
    int *list1 = node_list + start_addr[u-1] + 1;
    int len1 = list_len[u];

    int *list2 = node_list + start_addr[v-1] + 1;
    int len2 = list_len[v];

    /* 
     * Traversing both lists to find the common nodes. Each common node
     * will be counted as a triangle
     */
    while ( j < len1 && k < len2) {

        if (list1[j] == list2[k]) {
            count++;
            j++;
            k++;
        } else if (list1[j] < list2[k]) {
            j++;
        } else {
            k++;
        }
    }

    countArray[i] = count;
}


/*
 * Creating data structure which stores all the edges
 */
__global__ void createEdgeList(edge_tuple_t *edge_list, int *start_addr) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cuConstCounterParams.numNodes) {
        return;
    }

    if (i == 0) {
        return;
    }

    int *node_list = cuConstCounterParams.NodeList;
    int *list_len = cuConstCounterParams.ListLen;
    int start_index = start_addr[i-1] + 1;
    int *list = node_list + start_addr[i-1] + 1;
    int len = list_len[i];

    for (int j=0; j<len; j++) {
        edge_list[start_index].u = i;
        edge_list[start_index].v = list[j];
        start_index++;
    }
}


/*
 * Counts the number of triangles in the given graph. We first find out the
 * starting address of each list where list stores the neighbours of particular
 * node. We then create the list of all edges from the given nodes and their
 * neighbours.
 */
void
CudaTriangleCounter::countTriangles() {

    dim3 blockdim  = BLOCK_SIZE;
    dim3 griddim = (numEdges + BLOCK_SIZE)/BLOCK_SIZE;
    dim3 griddim1 = (numNodes + BLOCK_SIZE)/BLOCK_SIZE;
    int *countArray;
    int count;
    edge_tuple_t *edge_list;


    // Create CUDA events to keep track of the GPU time
    hipEvent_t Malloc3Start, Malloc3Stop;
    hipEventCreate(&Malloc3Start);
    hipEventCreate(&Malloc3Stop);
    // ----------------------------
    hipEvent_t GPUthrust1Start, GPUthrust1Stop;
    hipEventCreate(&GPUthrust1Start);
    hipEventCreate(&GPUthrust1Stop);
    // ----------------------------
    hipEvent_t GPUthrust2Start, GPUthrust2Stop;
    hipEventCreate(&GPUthrust2Start);
    hipEventCreate(&GPUthrust2Stop);
    // ----------------------------
    hipEvent_t GPUthrust3Start, GPUthrust3Stop;
    hipEventCreate(&GPUthrust3Start);
    hipEventCreate(&GPUthrust3Stop);
    // ----------------------------
    hipEvent_t Malloc4Start, Malloc4Stop;
    hipEventCreate(&Malloc4Start);
    hipEventCreate(&Malloc4Stop);
    // ----------------------------
    hipEvent_t CreateEdgesStart, CreateEdgesStop;
    hipEventCreate(&CreateEdgesStart);
    hipEventCreate(&CreateEdgesStop);
    // ----------------------------
    hipEvent_t Malloc5Start, Malloc5Stop;
    hipEventCreate(&Malloc5Start);
    hipEventCreate(&Malloc5Stop);
    // ----------------------------
    hipEvent_t TCountStart, TCountStop;
    hipEventCreate(&TCountStart);
    hipEventCreate(&TCountStop);
    // ----------------------------
    hipEvent_t GPUthrust4Start, GPUthrust4Stop;
    hipEventCreate(&GPUthrust4Start);
    hipEventCreate(&GPUthrust4Stop);
    // ----------------------------
    hipEvent_t GPUthrust5Start, GPUthrust5Stop;
    hipEventCreate(&GPUthrust5Start);
    hipEventCreate(&GPUthrust5Stop);
    // ----------------------------
    hipEvent_t Memcpy3Start, Memcpy3Stop;
    hipEventCreate(&Memcpy3Start);
    hipEventCreate(&Memcpy3Stop);
    // ----------------------------

    /* Calculating start address of each neighbour list */

    hipEventRecord(Malloc3Start, 0);
    hipMalloc(&cudaDeviceStartAddr, sizeof(int ) * numNodes);
    hipEventRecord(Malloc3Stop, 0);

    hipEventRecord(GPUthrust1Start, 0);
    thrust::device_ptr<int> dev_ptr1(cudaDeviceListLen);
    hipEventRecord(GPUthrust1Stop, 0);

    hipEventRecord(GPUthrust2Start, 0);
    thrust::device_ptr<int> output_ptr(cudaDeviceStartAddr);
    hipEventRecord(GPUthrust2Stop, 0);

    hipEventRecord(GPUthrust3Start, 0);
    thrust::inclusive_scan(dev_ptr1, dev_ptr1 + numNodes, output_ptr);
    hipEventRecord(GPUthrust3Stop, 0);


    /* Create a list of all edges present in the graph */
    hipEventRecord(Malloc4Start, 0);
    hipMalloc((void **)&edge_list, numEdges * sizeof(edge_tuple_t));
    hipEventRecord(Malloc4Stop, 0);

    hipEventRecord(CreateEdgesStart, 0);
    createEdgeList<<<griddim1, blockdim>>>(edge_list, cudaDeviceStartAddr);
    hipEventRecord(CreateEdgesStop, 0);


    hipDeviceSynchronize();


    hipEventRecord(Malloc5Start, 0);
    hipMalloc((void **)&countArray, numEdges * sizeof(int));
    hipEventRecord(Malloc5Stop, 0);


    /* Applyinf intersection rule on all edges to find number of triangles */
    hipEventRecord(TCountStart, 0);
    countTriangleKernel<<<griddim, blockdim>>>(countArray, edge_list, cudaDeviceStartAddr);
    hipEventRecord(TCountStop, 0);


    hipDeviceSynchronize();

    hipEventRecord(GPUthrust4Start, 0);
    thrust::device_ptr<int> dev_ptr(countArray);
    hipEventRecord(GPUthrust4Stop, 0);

    hipEventRecord(GPUthrust5Start, 0);
    thrust::inclusive_scan(dev_ptr, dev_ptr + numEdges, dev_ptr);
    hipEventRecord(GPUthrust5Stop, 0);

    hipEventRecord(Memcpy3Start, 0);
    hipMemcpy(&count, &countArray[numEdges-1], sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(Memcpy3Stop, 0);


    hipEventSynchronize(Malloc3Stop);
    float Malloc3ElapsedTime;
    hipEventElapsedTime(&Malloc3ElapsedTime, Malloc3Start, Malloc3Stop);
    cout << "Malloc3 copying took: " << Malloc3ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Malloc3Start);
    hipEventDestroy(Malloc3Stop);

    hipEventSynchronize(GPUthrust1Stop);
    float GPUthrust1ElapsedTime;
    hipEventElapsedTime(&GPUthrust1ElapsedTime, GPUthrust1Start, GPUthrust1Stop);
    cout << "GPUthrust1 copying took: " << GPUthrust1ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(GPUthrust1Start);
    hipEventDestroy(GPUthrust1Stop);

    hipEventSynchronize(GPUthrust2Stop);
    float GPUthrust2ElapsedTime;
    hipEventElapsedTime(&GPUthrust2ElapsedTime, GPUthrust2Start, GPUthrust2Stop);
    cout << "GPUthrust2 copying took: " << GPUthrust2ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(GPUthrust2Start);
    hipEventDestroy(GPUthrust2Stop);

    hipEventSynchronize(GPUthrust3Stop);
    float GPUthrust3ElapsedTime;
    hipEventElapsedTime(&GPUthrust3ElapsedTime, GPUthrust3Start, GPUthrust3Stop);
    cout << "GPUthrust3 copying took: " << GPUthrust3ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(GPUthrust3Start);
    hipEventDestroy(GPUthrust3Stop);

    hipEventSynchronize(Malloc4Stop);
    float Malloc4ElapsedTime;
    hipEventElapsedTime(&Malloc4ElapsedTime, Malloc4Start, Malloc4Stop);
    cout << "Malloc4 copying took: " << Malloc4ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Malloc4Start);
    hipEventDestroy(Malloc4Stop);

    hipEventSynchronize(CreateEdgesStop);
    float CreateEdgesElapsedTime;
    hipEventElapsedTime(&CreateEdgesElapsedTime, CreateEdgesStart, CreateEdgesStop);
    cout << "CreateEdges copying took: " << CreateEdgesElapsedTime << " milliseconds" << endl;
    hipEventDestroy(CreateEdgesStart);
    hipEventDestroy(CreateEdgesStop);

    hipEventSynchronize(Malloc5Stop);
    float Malloc5ElapsedTime;
    hipEventElapsedTime(&Malloc5ElapsedTime, Malloc5Start, Malloc5Stop);
    cout << "Malloc5 copying took: " << Malloc5ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Malloc5Start);
    hipEventDestroy(Malloc5Stop);

    hipEventSynchronize(TCountStop);
    float TCountElapsedTime;
    hipEventElapsedTime(&TCountElapsedTime, TCountStart, TCountStop);
    cout << "TCount  took: " << TCountElapsedTime << " milliseconds" << endl;
    hipEventDestroy(TCountStart);
    hipEventDestroy(TCountStop);

    hipEventSynchronize(GPUthrust4Stop);
    float GPUthrust4ElapsedTime;
    hipEventElapsedTime(&GPUthrust4ElapsedTime, GPUthrust4Start, GPUthrust4Stop);
    cout << "GPUthrust4 copying took: " << GPUthrust4ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(GPUthrust4Start);
    hipEventDestroy(GPUthrust4Stop);

    hipEventSynchronize(GPUthrust5Stop);
    float GPUthrust5ElapsedTime;
    hipEventElapsedTime(&GPUthrust5ElapsedTime, GPUthrust5Start, GPUthrust5Stop);
    cout << "GPUthrust5 copying took: " << GPUthrust5ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(GPUthrust5Start);
    hipEventDestroy(GPUthrust5Stop);

    hipEventSynchronize(Memcpy3Stop);
    float Memcpy3ElapsedTime;
    hipEventElapsedTime(&Memcpy3ElapsedTime, Memcpy3Start, Memcpy3Stop);
    cout << "Memcpy3 copying took: " << Memcpy3ElapsedTime << " milliseconds" << endl;
    hipEventDestroy(Memcpy3Start);
    hipEventDestroy(Memcpy3Stop);

    // Output the total GPU time to save in a csv file.
    cout << "Total GPU time: " << firstAllocation + Memcpy3ElapsedTime + GPUthrust5ElapsedTime + GPUthrust4ElapsedTime + TCountElapsedTime + Malloc5ElapsedTime +
            CreateEdgesElapsedTime + Malloc4ElapsedTime + GPUthrust3ElapsedTime + GPUthrust2ElapsedTime + GPUthrust1ElapsedTime +
            Malloc3ElapsedTime << " milliseconds" << endl;

    // printf("count %d\n", count);
}

